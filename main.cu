#include <iostream>
#include <hip/hip_runtime.h>

// __global__ means this function is going to be called by the CPU to run on the GPU
// __device__ means called by the GPU to run on the GPU
__global__ void test_func(int* value) {
    // Add ten to the pointer
    *value += 10;
}

int main() {
    std::cout << "Hello, World!" << std::endl;

    // Declare variables
    // "h_" is memory locations on the "host", i.e., CPU
    // "d_" are memory locations on the "device", i.e., GPU
    int test_number = 12;
    int* d_c;

    // cudaMalloc will allocate memory on the GPU
    // the first argument is the out parameter pointer, pointing to where the memory is on the GPU
    // do NOT dereference d_c on the CPU!
    hipMalloc((void**)&d_c, sizeof(int));

    // cudaMemcpy can copy memory to and from the CPU to the GPU
    // HostToDevice means copy from CPU to GPU
    // take data at h_c (a pointer which points to memory (an integer) on the CPU) and copy it to the destination memory address on the GPU (d_c)
    hipMemcpy(d_c, &test_number, sizeof(int), hipMemcpyHostToDevice);

    // Setup Block/Grid dimensions for the GPU
    // A "Block" is a chunk of threads and a "Grid" contains a certain number of "Blocks"
    dim3 grid_size(1); // (1,1,1), i.e., 1 block
    dim3 block_size(1); // (1,1,1), i.e., 1 thread per block

    void* args[] = {&d_c };
    // Launch the Kernel (i.e., execute on the GPU)
    hipLaunchKernel(reinterpret_cast<const void*>((void*)test_func),grid_size,block_size,args,0,NULL);

    // Copy the modified integer from the GPU back to the CPU
    hipMemcpy(&test_number, d_c, sizeof(int), hipMemcpyDeviceToHost);
    // Afterward, free GPU memory allocated
    hipFree(d_c);

    printf("The value received from the GPU is: %d",test_number);

    return 0;
}
